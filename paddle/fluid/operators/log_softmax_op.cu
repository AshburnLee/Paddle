#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <limits>
#include "paddle/fluid/operators/log_softmax_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"

namespace paddle {
namespace operators {

#define LAUNCH_WARP_FORWAR_COMPUTE(near_greater_power_of_two)                \
  case near_greater_power_of_two:                                            \
    ComputeForwardInWarp<T, double,                                          \
                         near_greater_power_of_two><<<blocks, threads, 0>>>( \
        dst, src, outer_size, dim_size, dim_size);                           \
    break;

template <typename T, int KernelWarpSize>
__device__ __forceinline__ void ReduceSumForWarpBatch(T &sum) {
#pragma unroll
  for (int offset = KernelWarpSize / 2; offset > 0; offset /= 2) {
    T sum_val = platform::CudaShuffleXorSync(0xFFFFFFFF, sum, offset);
    sum = sum + sum_val;
  }
}

template <typename T, int KernelWarpSize>
__device__ __forceinline__ void ReduceMaxForWarpBatch(T &sum) {
#pragma unroll
  for (int offset = KernelWarpSize / 2; offset > 0; offset /= 2) {
    T max_val = platform::CudaShuffleXorSync(0xFFFFFFFF, sum, offset);
    sum = max(sum, max_val);
  }
}

int GetNearGreaterPowerOfTwo(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) {
    ++log2_value;
  }
  return 1 << log2_value;
}

template <typename T, typename AccT, int NearGreaterPowerOfTwo>
__global__ void ComputeForwardInWarp(T *dst, const T *src, int batch_size,
                                     int stride, int element_count) {
  constexpr int near_greater_power_of_two = NearGreaterPowerOfTwo;
  constexpr int kernel_warp_size =
      (near_greater_power_of_two < 32) ? near_greater_power_of_two : 32;
  constexpr int warp_iter = near_greater_power_of_two / kernel_warp_size;
  int warp_id = blockDim.y * blockIdx.x + threadIdx.y;

  // effective_warp_id are binary values
  int effective_warp_id = batch_size - warp_id;
  if (effective_warp_id > 1) effective_warp_id = 1;

  int thread_in_warp_idx = threadIdx.x;

  // 1.read data from global memory to registers
  AccT elements[warp_iter];
  // set effective_element_count as the num of elements when warps do effectove
  // work
  // set effective_element_count as 0, when warps do ineffective work
  int effective_element_count = (effective_warp_id <= 0) ? 0 : element_count;
  for (int it = 0; it < warp_iter; ++it) {
    int element_index = thread_in_warp_idx + it * kernel_warp_size;
    if (element_index < effective_element_count) {
      elements[it] = static_cast<double>(
          src[warp_id * stride + thread_in_warp_idx + it * kernel_warp_size]);
    } else {
      elements[it] = -std::numeric_limits<AccT>::infinity();
    }
  }

  // 2.compute max_value. For each thread, loop all registers to find max
  AccT max_value;
  max_value = elements[0];
#pragma unroll
  for (int it = 1; it < warp_iter; ++it) {
    max_value = (max_value > elements[it]) ? max_value : elements[it];
  }
  ReduceMaxForWarpBatch<AccT, kernel_warp_size>(max_value);

  // 3.For each warp, accumulate all thread registers
  AccT sum = 0.0f;
#pragma unroll
  for (int it = 0; it < warp_iter; ++it) {
    sum += std::exp(elements[it] - max_value);
  }
  ReduceSumForWarpBatch<AccT, kernel_warp_size>(sum);

  // 4.store result.
  sum = std::log(sum);
#pragma unroll
  for (int it = 0; it < warp_iter; ++it) {
    int element_index = thread_in_warp_idx + it * kernel_warp_size;
    if (element_index < element_count) {
      dst[warp_id * stride + thread_in_warp_idx + it * kernel_warp_size] =
          elements[it] - max_value - sum;
    } else {
      break;
    }
  }
}

template <typename T>
void LaunchSoftmaxForwardForLastAxis(T *dst, const T *src, int dim_size,
                                     int outer_size) {
  int threads_per_block = 128;
  int near_greater_power_of_two = GetNearGreaterPowerOfTwo(dim_size);
  int kernel_warp_size =
      (near_greater_power_of_two < 32) ? near_greater_power_of_two : 32;
  int warps_per_block = (threads_per_block / kernel_warp_size);
  int blocks = (outer_size + warps_per_block - 1) / warps_per_block;
  dim3 threads(kernel_warp_size, warps_per_block, 1);

  switch (near_greater_power_of_two) {
    LAUNCH_WARP_FORWAR_COMPUTE(1);
    LAUNCH_WARP_FORWAR_COMPUTE(2);
    LAUNCH_WARP_FORWAR_COMPUTE(4);     // dim_size: 3~4
    LAUNCH_WARP_FORWAR_COMPUTE(8);     // dim_size: 5~8
    LAUNCH_WARP_FORWAR_COMPUTE(16);    // dim_size: 9~16
    LAUNCH_WARP_FORWAR_COMPUTE(32);    // dim_size: 17~32
    LAUNCH_WARP_FORWAR_COMPUTE(64);    // dim_size: 33~64
    LAUNCH_WARP_FORWAR_COMPUTE(128);   // dim_size 65~128
    LAUNCH_WARP_FORWAR_COMPUTE(256);   // dim_size 129~256
    LAUNCH_WARP_FORWAR_COMPUTE(512);   // dim_size 257~512
    LAUNCH_WARP_FORWAR_COMPUTE(1024);  // dim_size 513~1024

    default:
      break;
  }
}

template <typename T>
class LogSoftmaxKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    const auto *x = context.Input<framework::Tensor>("X");
    auto *out = context.Output<framework::Tensor>("Out");
    const auto *input_data = x->data<T>();
    auto *output_data = out->mutable_data<T>(context.GetPlace());

    PADDLE_ENFORCE_GT(x->numel(), 0, platform::errors::InvalidArgument(
                                         "Expected number of elements > 0. But "
                                         "received number of elements is %d.",
                                         x->numel()));
    const int rank = x->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);

    int dim_size = x->dims()[axis];
    int inner_size = 1;
    for (int i = axis + 1; i < x->dims().size(); i++) {
      inner_size *= x->dims()[i];
    }
    int outer_size = SizeToAxis(axis, x->dims());

    if (inner_size == 1 && dim_size <= 1024 && dim_size * sizeof(T) <= 4096) {
      // execute CUDA kernel
      LaunchSoftmaxForwardForLastAxis<T>(output_data, input_data, dim_size,
                                         outer_size);
    } else {
      // execute Eigen kernel
      LogSoftmaxFunctor<platform::CUDADeviceContext, T>()(
          context.template device_context<platform::CUDADeviceContext>(), x,
          out, axis);
    }
  }
};

}  // operators
}  // paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    log_softmax, ops::LogSoftmaxKernel<plat::CUDADeviceContext, float>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, double>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    log_softmax_grad, ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, float>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, double>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, plat::float16>);
