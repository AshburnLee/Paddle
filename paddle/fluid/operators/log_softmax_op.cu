#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <limits>
#include "paddle/fluid/operators/log_softmax_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"

namespace paddle {
namespace operators {

#define WARP_SIZE 32

#define LAUNCH_SOFTMAX_WARP_FORWARD(L2E)                                   \
  case L2E:                                                                \
    WarpLogSoftmaxForward<T, double, L2E><<<blocks, threads, 0>>>(         \
        dst, src, batch_count, softmax_elements_stride, softmax_elements); \
    break;

int LogTwoCeil(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) ++log2_value;
  return log2_value;
}

template <typename T, int NumBatch, int KernelWarpSize>
__device__ __forceinline__ void ReduceSumForWarpBatch(T* sum) {
#pragma unroll
  for (int offset = KernelWarpSize / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < NumBatch; ++i) {
      T sum_val = platform::CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = sum[i] + sum_val;
    }
  }
}

template <typename T, int NumBatch, int KernelWarpSize>
__device__ __forceinline__ void ReduceMaxForWarpBatch(T* sum) {
#pragma unroll
  for (int offset = KernelWarpSize / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < NumBatch; ++i) {
      T max_val = platform::CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = max(sum[i], max_val);
    }
  }
}

template <typename T, typename AccT, int log2_elements>
__global__ void WarpLogSoftmaxForward(T* dst, const T* src, int batch_size,
                                      int stride, int element_count) {
  constexpr int next_power_of_two = 1 << log2_elements;
  constexpr int kernel_warp_size =
      (next_power_of_two < WARP_SIZE) ? next_power_of_two : WARP_SIZE;
  constexpr int warp_iterations = next_power_of_two / kernel_warp_size;
  constexpr int num_batch = (next_power_of_two <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * num_batch;
  int local_batches = batch_size - first_batch;
  if (local_batches > num_batch) local_batches = num_batch;

  int local_idx = threadIdx.x;
  src += first_batch * stride + local_idx;
  dst += first_batch * stride + local_idx;

  // 1.load data from global memory
  AccT elements[num_batch][warp_iterations];
  int idx = threadIdx.x + blockDim.x * threadIdx.y;

  for (int i = 0; i < num_batch; ++i) {
    int batch_element_count = (i >= local_batches) ? 0 : element_count;
    for (int it = 0; it < warp_iterations; ++it) {
      int element_index = local_idx + it * kernel_warp_size;
      if (element_index < batch_element_count) {
        elements[i][it] =
            static_cast<double>(src[i * element_count + it * kernel_warp_size]);
      } else {
        elements[i][it] = -std::numeric_limits<AccT>::infinity();
      }
    }
  }

  // 2.compute max_value
  AccT max_value[num_batch];
#pragma unroll
  for (int i = 0; i < num_batch; ++i) {
    max_value[i] = elements[i][0];
#pragma unroll
    for (int it = 1; it < warp_iterations; ++it) {
      max_value[i] =
          (max_value[i] > elements[i][it]) ? max_value[i] : elements[i][it];
    }
  }
  ReduceMaxForWarpBatch<AccT, num_batch, kernel_warp_size>(max_value);

  AccT sum[num_batch]{0.0f};
#pragma unroll
  for (int i = 0; i < num_batch; ++i) {
#pragma unroll
    for (int it = 0; it < warp_iterations; ++it) {
      sum[i] += std::exp(elements[i][it] - max_value[i]);
    }
  }
  ReduceSumForWarpBatch<AccT, num_batch, kernel_warp_size>(sum);

// 3.store result
#pragma unroll
  for (int i = 0; i < num_batch; ++i) {
    if (i >= local_batches) break;
    sum[i] = std::log(sum[i]);
#pragma unroll
    for (int it = 0; it < warp_iterations; ++it) {
      int element_index = local_idx + it * kernel_warp_size;
      if (element_index < element_count) {
        dst[i * element_count + it * kernel_warp_size] =
            elements[i][it] - max_value[i] - sum[i];
      } else {
        break;
      }
    }
  }
}

template <typename T>
void LaunchSoftmaxForwardForLastAxis(T* dst, const T* src, int softmax_elements,
                                     int softmax_elements_stride,
                                     int batch_count) {
  int log2_elements = LogTwoCeil(softmax_elements);
  const int next_power_of_two = 1 << log2_elements;
  int warp_size =
      (next_power_of_two < WARP_SIZE) ? next_power_of_two : WARP_SIZE;
  int batches_per_warp = (next_power_of_two <= 128) ? 2 : 1;

  // use 128 threads per block to maximimize gpu utilization
  constexpr int threads_per_block = 128;
  int warps_per_block = (threads_per_block / warp_size);
  int batches_per_block = warps_per_block * batches_per_warp;
  int blocks = (batch_count + batches_per_block - 1) / batches_per_block;
  dim3 threads(warp_size, warps_per_block, 1);

  switch (log2_elements) {
    LAUNCH_SOFTMAX_WARP_FORWARD(0);   // 1
    LAUNCH_SOFTMAX_WARP_FORWARD(1);   // 2
    LAUNCH_SOFTMAX_WARP_FORWARD(2);   // 4
    LAUNCH_SOFTMAX_WARP_FORWARD(3);   // 8
    LAUNCH_SOFTMAX_WARP_FORWARD(4);   // 16
    LAUNCH_SOFTMAX_WARP_FORWARD(5);   // 32
    LAUNCH_SOFTMAX_WARP_FORWARD(6);   // 64
    LAUNCH_SOFTMAX_WARP_FORWARD(7);   // 128
    LAUNCH_SOFTMAX_WARP_FORWARD(8);   // 256
    LAUNCH_SOFTMAX_WARP_FORWARD(9);   // 512
    LAUNCH_SOFTMAX_WARP_FORWARD(10);  // 1024
    default:
      break;
  }
}

template <typename T>
class LogSoftmaxKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    const auto* X = context.Input<framework::Tensor>("X");
    auto* Out = context.Output<framework::Tensor>("Out");
    const auto* input_data = X->data<T>();
    auto* output_data = Out->mutable_data<T>(context.GetPlace());

    PADDLE_ENFORCE_GT(X->numel(), 0, platform::errors::InvalidArgument(
                                         "Expected number of elements > 0. But "
                                         "received number of elements is %d.",
                                         X->numel()));
    const int rank = X->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);

    int dim_size = X->dims()[axis];
    int inner_size = 1;
    for (int i = axis + 1; i < X->dims().size(); i++)
      inner_size *= X->dims()[i];
    int outer_size = 1;
    outer_size = SizeToAxis(axis, X->dims());

    if (inner_size == 1 && dim_size <= 1024 && dim_size * sizeof(T) <= 4096) {
      // execute CUDA kernel
      LaunchSoftmaxForwardForLastAxis<T>(output_data, input_data, dim_size,
                                         dim_size, outer_size);
    } else {
      // execute Eigen kernel
      LogSoftmaxFunctor<platform::CUDADeviceContext, T>()(
          context.template device_context<platform::CUDADeviceContext>(), X,
          Out, axis);
    }
  }
};

}  // operators
}  // paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    log_softmax, ops::LogSoftmaxKernel<plat::CUDADeviceContext, float>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, double>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    log_softmax_grad, ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, float>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, double>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, plat::float16>);
